#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>

#ifndef SHA256_CUH
#define SHA256_CUH

// __constant__ array for device-side K values
__constant__ static const uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

// Host-side equivalent of K for use in host functions
static const uint32_t K_host[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

// Custom device-compatible string copy function
__device__ void cuda_strcpy(char *dest, const char *src, size_t max_length)
{
    size_t i = 0;
    while (src[i] && i < max_length - 1)
    { // Ensure we don't exceed max_length
        dest[i] = src[i];
        i++;
    }
    dest[i] = '\0'; // Null terminate
}

// Custom device-compatible string concatenate function
__device__ void cuda_strcat(char *dest, const char *src)
{
    while (*dest)
        dest++; // Move pointer to the end of dest
    while (*src)
    {
        *dest++ = *src++;
    }
    *dest = '\0'; // Null terminate
}

// Custom device-compatible string length function
__device__ size_t cuda_strlen(const char *str)
{
    size_t len = 0;
    while (*str++)
        len++;
    return len;
}

class SHA256
{

public:
    __device__ __host__ SHA256()
    {
        m_blocklen = 0;
        m_bitlen = 0;
        m_state[0] = 0x6a09e667;
        m_state[1] = 0xbb67ae85;
        m_state[2] = 0x3c6ef372;
        m_state[3] = 0xa54ff53a;
        m_state[4] = 0x510e527f;
        m_state[5] = 0x9b05688c;
        m_state[6] = 0x1f83d9ab;
        m_state[7] = 0x5be0cd19;
    }

    __device__ __host__ void update(const uint8_t *data, size_t length)
    {
        for (size_t i = 0; i < length; i++)
        {
            m_data[m_blocklen++] = data[i];
            if (m_blocklen == 64)
            {
                transform();
                m_bitlen += 512;
                m_blocklen = 0;
            }
        }
    }

    __device__ __host__ void digest(uint8_t *hash)
    {
        pad();
        revert(hash);
    }

private:
    uint8_t m_data[64];
    uint32_t m_blocklen;
    uint64_t m_bitlen;
    uint32_t m_state[8]; // A, B, C, D, E, F, G, H

    __device__ __host__ static uint32_t rotr(uint32_t x, uint32_t n)
    {
        return (x >> n) | (x << (32 - n));
    }

    __device__ __host__ static uint32_t choose(uint32_t e, uint32_t f, uint32_t g)
    {
        return (e & f) ^ (~e & g);
    }

    __device__ __host__ static uint32_t majority(uint32_t a, uint32_t b, uint32_t c)
    {
        return (a & (b | c)) | (b & c);
    }

    __device__ __host__ static uint32_t sig0(uint32_t x)
    {
        return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
    }

    __device__ __host__ static uint32_t sig1(uint32_t x)
    {
        return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
    }

    __device__ __host__ void transform()
    {
        uint32_t maj, xorA, ch, xorE, sum, newA, newE, m[64];
        uint32_t state[8];

        // Select K array based on whether we're on the device or host
#ifdef __CUDA_ARCH__
        const uint32_t *k_array = K; // For device
#else
        const uint32_t *k_array = K_host; // For host
#endif

        // Process the message schedule array (W)
        for (uint8_t i = 0, j = 0; i < 16; i++, j += 4)
        {
            m[i] = (m_data[j] << 24) | (m_data[j + 1] << 16) | (m_data[j + 2] << 8) | m_data[j + 3];
        }

        for (uint8_t k = 16; k < 64; k++)
        {
            m[k] = sig1(m[k - 2]) + m[k - 7] + sig0(m[k - 15]) + m[k - 16];
        }

        // Initialize state array with the current hash values
        for (uint8_t i = 0; i < 8; i++)
        {
            state[i] = m_state[i];
        }

        // Main compression loop
        for (uint8_t i = 0; i < 64; i++)
        {
            maj = majority(state[0], state[1], state[2]);
            xorA = rotr(state[0], 2) ^ rotr(state[0], 13) ^ rotr(state[0], 22);

            ch = choose(state[4], state[5], state[6]);
            xorE = rotr(state[4], 6) ^ rotr(state[4], 11) ^ rotr(state[4], 25);

            sum = m[i] + k_array[i] + state[7] + ch + xorE;
            newA = xorA + maj + sum;
            newE = state[3] + sum;

            state[7] = state[6];
            state[6] = state[5];
            state[5] = state[4];
            state[4] = newE;
            state[3] = state[2];
            state[2] = state[1];
            state[1] = state[0];
            state[0] = newA;
        }

        // Add the compressed chunk to the current hash value
        for (uint8_t i = 0; i < 8; i++)
        {
            m_state[i] += state[i];
        }
    }

    __device__ __host__ void pad()
    {
        uint64_t i = m_blocklen;
        uint8_t end = m_blocklen < 56 ? 56 : 64;

        m_data[i++] = 0x80; // Append 1 bit followed by zeros
        while (i < end)
        {
            m_data[i++] = 0x00;
        }

        if (m_blocklen >= 56)
        {
            transform();
            memset(m_data, 0, 56);
        }

        m_bitlen += m_blocklen * 8;
        m_data[63] = m_bitlen;
        m_data[62] = m_bitlen >> 8;
        m_data[61] = m_bitlen >> 16;
        m_data[60] = m_bitlen >> 24;
        m_data[59] = m_bitlen >> 32;
        m_data[58] = m_bitlen >> 40;
        m_data[57] = m_bitlen >> 48;
        m_data[56] = m_bitlen >> 56;
        transform();
    }

    __device__ __host__ void revert(uint8_t *hash)
    {
        for (uint8_t i = 0; i < 4; i++)
        {
            for (uint8_t j = 0; j < 8; j++)
            {
                hash[i + (j * 4)] = (m_state[j] >> (24 - i * 8)) & 0x000000ff;
            }
        }
    }
};

#endif

__device__ void computeHash(const char *password, uint8_t *hashOutput)
{
    // Create an instance of SHA256
    SHA256 sha256;

    // Hash the password
    sha256.update((const uint8_t *)password, cuda_strlen(password));

    // Get the resulting hash
    sha256.digest(hashOutput);
}

__device__ void hashToHex(const uint8_t *hash, char *hexOutput)
{
    const char hexDigits[] = "0123456789abcdef";
    for (int i = 0; i < 32; ++i)
    {
        hexOutput[i * 2] = hexDigits[(hash[i] >> 4) & 0xF]; // High nibble
        hexOutput[i * 2 + 1] = hexDigits[hash[i] & 0xF];    // Low nibble
    }
    hexOutput[64] = '\0'; // Null-terminate the string
}

__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
const int charset_size = 62; // Length of charset
const size_t password_length = 6;
const size_t hash_length = 64;
const size_t salt_length = 16;

__device__ void generate_password(long long idx, char *password)
{
    for (int i = 0; i < password_length; ++i)
    {
        password[i] = charset[idx % charset_size];
        idx /= charset_size;
    }
    password[password_length] = '\0'; // Null-terminate the string
}

__device__ bool custom_strcmp(const char *a, const char *b)
{
    for (int i = 0; i < password_length; ++i)
    {
        if (a[i] != b[i])
        {
            return false;
        }
    }
    return true;
}

__device__ bool compareUint8Arrays(const uint8_t* array1, const uint8_t* array2, size_t length) {
    for (size_t i = 0; i < length; ++i) {
        if (array1[i] != array2[i]) {
            return false; // Arrays differ at this position
        }
    }
    return true; // Arrays are identical
}

void hexToBytes(const char *hexString, uint8_t *byteArray)
{
    for (size_t i = 0; i < 32; ++i)
    {
        sscanf(hexString + 2 * i, "%2hhx", &byteArray[i]);
    }
}

std::string bytesToHex(const uint8_t *byteArray, size_t length)
{
    std::ostringstream oss;
    for (size_t i = 0; i < length; ++i)
    {
        oss << std::hex << std::setw(2) << std::setfill('0') << static_cast<int>(byteArray[i]);
    }
    return oss.str();
}


__global__ void find_password(long long start, long long end, int *found, long long *result_index, const char *target_salt, const uint8_t *target_hash)
{
    long long idx = blockIdx.x * blockDim.x + threadIdx.x + start;
    uint8_t sha256_uint8[32];

    if (idx < end)
    {
        char password[password_length + 1];
        char combined_salt[salt_length + password_length + 1];
        generate_password(idx, password);

        cuda_strcpy(combined_salt, target_salt, salt_length + 1);
        cuda_strcat(combined_salt, password);

        // Create an instance of SHA256
        SHA256 sha256;

        // Hash the password
        sha256.update((const uint8_t *)combined_salt, cuda_strlen(combined_salt));

        // Get the resulting hash
        sha256.digest(sha256_uint8);

        if (compareUint8Arrays(target_hash, sha256_uint8, 32))
        {
            if (atomicExch(found, 1) == 0)
            {
                // Only the first thread to find the password will update result_index
                *result_index = idx;
            }
        }
    }
}

__global__ void find_password_optimized(long long start, long long end, int *found, long long *result_index, const char *target_salt, const uint8_t *target_hash) {
    
    long long idx = blockIdx.x * blockDim.x + threadIdx.x + start;
    __shared__ uint8_t shared_target_hash[32]; // Use shared memory for target hash

    if (threadIdx.x < 32) {
        shared_target_hash[threadIdx.x] = target_hash[threadIdx.x];
    }
    __syncthreads();

    if (idx < end) {
        char password[password_length + 1];
        char combined_salt[salt_length + password_length + 1];
        generate_password(idx, password);

        cuda_strcpy(combined_salt, target_salt, salt_length + 1);
        cuda_strcat(combined_salt, password);

        // Create an instance of SHA256
        SHA256 sha256;

        // Hash the password
        sha256.update((const uint8_t *)combined_salt, cuda_strlen(combined_salt));

        // Get the resulting hash
        uint8_t sha256_uint8[32];
        sha256.digest(sha256_uint8);

        if (compareUint8Arrays(shared_target_hash, sha256_uint8, 32)) {
            if (atomicExch(found, 1) == 0) {
                *result_index = idx;
            }
        }
    }
}



int main()
{
    // Open the input file
    std::ifstream infile("in.txt");
    if (!infile) {
        std::cerr << "Unable to open file in.txt";
        return 1;
    }

    std::string line;

    while (std::getline(infile, line)) {
            std::string salt_hex_string = line.substr(0, 16); // First 24 characters for salt (12 bytes)
            std::string target_hash_string = line.substr(18, 66); // Next 64 characters for target hash (32 bytes)

            printf("Salt: %s\n", salt_hex_string.c_str());
            printf("Target Hash: %s\n", target_hash_string.c_str());

            const char *target_salt = salt_hex_string.c_str();
            const char *target_hash_hex = target_hash_string.c_str();
            uint8_t target_hash[32];

            // Convert the target hash from hex string to byte array
            hexToBytes(target_hash_hex, target_hash);


            long long total_passwords = 62LL * 62 * 62 * 62 * 62 * 62; // 62^6 with explicit long long
            long long blockSize = 256;                                 // Number of threads per block
            long long passwords_per_batch = 1000000;                   // Number of passwords to process in one batch
            long long num_batches = (total_passwords + passwords_per_batch - 1) / passwords_per_batch;

            char *d_target_salt;
            uint8_t *d_target_hash;
            char *d_salt;

            int *d_found;
            int found = 0;
            long long *d_result_index;

            hipMalloc(&d_found, sizeof(int));
            hipMalloc(&d_salt, (salt_length + 1) * sizeof(char));
            hipMalloc(&d_target_hash, 32 * sizeof(uint8_t));
            hipMalloc(&d_result_index, sizeof(long long));

            hipMemcpy(d_found, &found, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_salt, target_salt, (salt_length + 1) * sizeof(char), hipMemcpyHostToDevice);
            hipMemcpy(d_target_hash, target_hash, 32 * sizeof(uint8_t), hipMemcpyHostToDevice);

            // Start timing
            auto start_time = std::chrono::high_resolution_clock::now();

            for (long long batch = 0; batch < num_batches; ++batch)
            {
                long long start = batch * passwords_per_batch;
                long long end = min(start + passwords_per_batch, total_passwords);

                // Calculate number of blocks needed for this batch
                long long numBlocks = (end - start + blockSize - 1) / blockSize;

                // Launch kernel for the current batch
                find_password<<<numBlocks, blockSize>>>(start, end, d_found, d_result_index, d_salt, d_target_hash);
                hipError_t err = hipGetLastError();
                if (err != hipSuccess) {
                    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
                }
                hipDeviceSynchronize();

                // Copy results back to host
                long long result_index;
                hipMemcpy(&found, d_found, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&result_index, d_result_index, sizeof(long long), hipMemcpyDeviceToHost);

                if (found == 1)
                {
                    std::cout << "Password found at index: " << result_index << "\n";
                    break; // Exit loop if password is found
                }
            }

            // End timing
            auto end_time = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> elapsed_seconds = end_time - start_time;

            // Calculate GH/s
            double hashes_per_second = total_passwords / elapsed_seconds.count();
            double gigahashes_per_second = hashes_per_second / 1e9;
            std::cout << "Performance: " << gigahashes_per_second << " GH/s" << std::endl;

            // Free device memory
            hipFree(d_found);
            hipFree(d_result_index);
    }
    infile.close();
    return 0;
}
