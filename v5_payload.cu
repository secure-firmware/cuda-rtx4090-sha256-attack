#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>

#ifndef SHA256_CUH
#define SHA256_CUH

// __constant__ array for device-side K values
__constant__ static const uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__constant__ char d_target_salt[16 + 1];
__constant__ uint8_t d_target_hash[32];

// Host-side equivalent of K for use in host functions
static const uint32_t K_host[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

// Custom device-compatible string copy function
__device__ void cuda_strcpy(char *dest, const char *src, size_t max_length)
{
    size_t i = 0;
    while (src[i] && i < max_length - 1)
    { // Ensure we don't exceed max_length
        dest[i] = src[i];
        i++;
    }
    dest[i] = '\0'; // Null terminate
}

// Custom device-compatible string concatenate function
__device__ void cuda_strcat(char *dest, const char *src)
{
    while (*dest)
        dest++; // Move pointer to the end of dest
    while (*src)
    {
        *dest++ = *src++;
    }
    *dest = '\0'; // Null terminate
}

// Custom device-compatible string length function
__device__ size_t cuda_strlen(const char *str)
{
    size_t len = 0;
    while (*str++)
        len++;
    return len;
}

class SHA256
{
public:
    uint32_t m_state[8]; // A, B, C, D, E, F, G, H
    uint32_t m_saltState[8]; // State after salt processing
    uint64_t m_saltBitlen; // Bit length after salt processing
    uint32_t m_saltBlocklen; // Block length after salt processing

    __device__ __host__ SHA256()
    {
        reset();
    }

    __device__ __host__ void reset()
    {
        m_blocklen = 0;
        m_bitlen = 0;
        m_state[0] = 0x6a09e667;
        m_state[1] = 0xbb67ae85;
        m_state[2] = 0x3c6ef372;
        m_state[3] = 0xa54ff53a;
        m_state[4] = 0x510e527f;
        m_state[5] = 0x9b05688c;
        m_state[6] = 0x1f83d9ab;
        m_state[7] = 0x5be0cd19;
    }

    __device__ __host__ void initWithSalt(const uint8_t *salt, size_t salt_length)
    {
        reset();
        update(salt, salt_length);

        //Store the state after processing with the salt
        for (int i = 0; i < 8; i++)
        {
            m_saltState[i] = m_state[i];
        }
        m_saltBitlen = m_bitlen;
        m_saltBlocklen = m_blocklen;
    }

    __device__ __host__ void resetToSaltState()
    {
        for (int i = 0; i < 8; i++)
        {
            m_state[i] = m_saltState[i];
        }
        m_bitlen = m_saltBitlen;
        m_blocklen = m_saltBlocklen;
    }

    __device__ __host__ void update(const uint8_t *data, size_t length)
    {
        for (size_t i = 0; i < length; i++)
        {
            m_data[m_blocklen++] = data[i];
            if (m_blocklen == 64)
            {
                transform();
                m_bitlen += 512;
                m_blocklen = 0;
            }
        }
    }

    __device__ __host__ void digest(uint8_t *hash)
    {
        pad();
        revert(hash);
    }

private:
    uint8_t m_data[64];
    uint32_t m_blocklen;
    uint64_t m_bitlen;

    __device__ __host__ static uint32_t rotr(uint32_t x, uint32_t n)
    {
        return (x >> n) | (x << (32 - n));
    }

    __device__ __host__ static uint32_t choose(uint32_t e, uint32_t f, uint32_t g)
    {
        return (e & f) ^ (~e & g);
    }

    __device__ __host__ static uint32_t majority(uint32_t a, uint32_t b, uint32_t c)
    {
        return (a & (b | c)) | (b & c);
    }

    __device__ __host__ static uint32_t sig0(uint32_t x)
    {
        return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
    }

    __device__ __host__ static uint32_t sig1(uint32_t x)
    {
        return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
    }

    __device__ __host__ void transform()
    {
        uint32_t maj, xorA, ch, xorE, sum, newA, newE, m[64];
        uint32_t state[8];

        #ifdef __CUDA_ARCH__
        const uint32_t *k_array = K;
        #else
        const uint32_t *k_array = K_host;
        #endif

        // Unroll the first loop for processing the message schedule array
        #pragma unroll 16
        for (uint8_t i = 0, j = 0; i < 16; i++, j += 4)
        {
            m[i] = (m_data[j] << 24) | (m_data[j + 1] << 16) | (m_data[j + 2] << 8) | m_data[j + 3];
        }

        // Unroll the second loop for the message schedule array
        #pragma unroll 48
        for (uint8_t k = 16; k < 64; k++)
        {
            m[k] = sig1(m[k - 2]) + m[k - 7] + sig0(m[k - 15]) + m[k - 16];
        }

        // Initialize state array with the current hash values
        #pragma unroll 8
        for (uint8_t i = 0; i < 8; i++)
        {
            state[i] = m_state[i];
        }

        // Main compression loop - fully unroll
        #pragma unroll 64
        for (uint8_t i = 0; i < 64; i++)
        {
            maj = majority(state[0], state[1], state[2]);
            xorA = rotr(state[0], 2) ^ rotr(state[0], 13) ^ rotr(state[0], 22);

            ch = choose(state[4], state[5], state[6]);
            xorE = rotr(state[4], 6) ^ rotr(state[4], 11) ^ rotr(state[4], 25);

            sum = m[i] + k_array[i] + state[7] + ch + xorE;
            newA = xorA + maj + sum;
            newE = state[3] + sum;

            state[7] = state[6];
            state[6] = state[5];
            state[5] = state[4];
            state[4] = newE;
            state[3] = state[2];
            state[2] = state[1];
            state[1] = state[0];
            state[0] = newA;
        }

        // Add the compressed chunk to the current hash value
        #pragma unroll 8
        for (uint8_t i = 0; i < 8; i++)
        {
            m_state[i] += state[i];
        }
    }

    __device__ __host__ void pad()
    {
        uint64_t i = m_blocklen;
        uint8_t end = m_blocklen < 56 ? 56 : 64;

        m_data[i++] = 0x80; // Append 1 bit followed by zeros
        while (i < end)
        {
            m_data[i++] = 0x00;
        }

        if (m_blocklen >= 56)
        {
            transform();
            memset(m_data, 0, 56);
        }

        m_bitlen += m_blocklen * 8;
        m_data[63] = m_bitlen;
        m_data[62] = m_bitlen >> 8;
        m_data[61] = m_bitlen >> 16;
        m_data[60] = m_bitlen >> 24;
        m_data[59] = m_bitlen >> 32;
        m_data[58] = m_bitlen >> 40;
        m_data[57] = m_bitlen >> 48;
        m_data[56] = m_bitlen >> 56;
        transform();
    }

    __device__ __host__ void revert(uint8_t *hash)
    {
        for (uint8_t i = 0; i < 4; i++)
        {
            for (uint8_t j = 0; j < 8; j++)
            {
                hash[i + (j * 4)] = (m_state[j] >> (24 - i * 8)) & 0x000000ff;
            }
        }
    }
};

#endif

__device__ void computeHash(const char *password, uint8_t *hashOutput)
{
    // Create an instance of SHA256
    SHA256 sha256;

    // Hash the password
    sha256.update((const uint8_t *)password, cuda_strlen(password));

    // Get the resulting hash
    sha256.digest(hashOutput);
}


__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
const int charset_size = 62; // Length of charset
const size_t password_length = 6;
const size_t salt_length = 16;


void hexToBytes(const char *hexString, uint8_t *byteArray)
{
    for (size_t i = 0; i < 32; ++i)
    {
        sscanf(hexString + 2 * i, "%2hhx", &byteArray[i]);
    }
}

__device__ void generate_password(long long idx, char *password)
{
    for (int i = 0; i < password_length; ++i)
    {
        password[i] = charset[idx % charset_size];
        idx /= charset_size;
    }
    password[password_length] = '\0'; // Null-terminate the string
}

__device__ bool compareUint8Arrays(const uint8_t* array1, const uint8_t* array2, size_t length) {
    for (size_t i = 0; i < length; ++i) {
        if (array1[i] != array2[i]) {
            return false; // Arrays differ at this position
        }
    }
    return true; // Arrays are identical
}

__global__ void find_passwords_optimized_multi(
    const char* salt,                // Input: Salt for hash
    const uint8_t* target_hashes,    // Input: Array of target hashes
    int num_target_hashes,           // Input: Number of target hashes
    int* found_flags,                // Output: Flags indicating which hashes are found
    long long* result_indices,       // Output: Indices of found passwords
    unsigned char* checked_bitmap,   // Input/Output: Bitmap to track checked passwords
    unsigned long long* global_start_index,   // Input/Output: Global counter for password indices
    int batch_size                  // Input: Number of passwords each thread processes
) {
    // Shared memory for storing the initial SHA256 state with salt
    __shared__ SHA256 shared_sha256;

    // Initialize the SHA256 state with salt (only first thread in block)
    if (threadIdx.x == 0) {
        shared_sha256.initWithSalt((const uint8_t*)salt, salt_length);
    }

    __syncthreads(); // Ensure all threads have access to initialized shared memory

    // Get the starting index for this thread's batch of passwords
    long long thread_start_index = atomicAdd((unsigned long long*)global_start_index, (unsigned long long)batch_size);
    uint8_t hash[32]; // Buffer to store computed hash

    // Initialize SHA256 object for this thread
    SHA256 sha256 = shared_sha256;

    // Process batch_size number of passwords
    for (int i = 0; i < batch_size; i++) {
        long long idx = thread_start_index + i;
        
        // Check if this password index has already been processed
        int byte_index = idx / 8;
        int bit_index = idx % 8;
        unsigned int mask = 1U << bit_index;
        unsigned char old = atomicOr((unsigned int*)&checked_bitmap[byte_index], (unsigned int)mask);
        if (old & mask) {
            continue;  // Skip already checked passwords
        }

        // Generate password for this index
        char password[password_length + 1];
        generate_password(idx, password);

        // Compute hash for the password
        sha256.resetToSaltState();
        sha256.update((const uint8_t*)password, password_length);
        sha256.digest(hash);

        // Compare with all target hashes
        for (int j = 0; j < num_target_hashes; j++) {
            if (!found_flags[j] && compareUint8Arrays(hash, target_hashes + j * 32, 32)) {
                // Atomically set the found flag and store the result index
                int old = atomicExch(&found_flags[j], 1);
                if (old == 0) {
                    result_indices[j] = idx;
                }
            }
        }
    }
}



int main()
{

    int maxThreadsPerBlock;
    int maxBlocksPerSM;
    int numSMs;

    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&maxBlocksPerSM, hipDeviceAttributeMaxBlocksPerMultiprocessor, 0);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    const int NUM_BLOCK_SIZES = 5;
    int blockSizes[NUM_BLOCK_SIZES] = {64, 128, 256, 512, 1024};

    for (int i = 0; i < NUM_BLOCK_SIZES; i++) {
        int blockSize = blockSizes[i];
        int numBlocks;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, find_passwords_optimized_multi, blockSize, 0);
        float occupancy = (float)(numBlocks * blockSize) / maxThreadsPerBlock;
        std::cout << "Block size: " << blockSize << ", Occupancy: " << occupancy * 100 << "%" << std::endl;
    }

    // Open the input file
    std::ifstream infile("in.txt");
    if (!infile) {
        std::cerr << "Unable to open file in.txt";
        return 1;
    }

    std::string line;

    while (std::getline(infile, line)) {
            std::string salt_hex_string = line.substr(0, 16); // First 24 characters for salt (12 bytes)
            std::string target_hash_string = line.substr(18, 82); // Next 64 characters for target hash (32 bytes)

            printf("Salt: %s\n", salt_hex_string.c_str());
            printf("Target Hash: %s\n", target_hash_string.c_str());

            const char *target_salt = salt_hex_string.c_str();
            const char *target_hash_hex = target_hash_string.c_str();
            uint8_t target_hash[32];

            // Convert the target hash from hex string to byte array
            hexToBytes(target_hash_hex, target_hash);


            unsigned long long total_passwords = 62ULL * 62 * 62 * 62 * 62 * 62; // 62^6
            int blockSize = 256; // Adjust based on your GPU capabilities
            int batch_size = 100; // Adjust based on optimal performance
            int numBlocks = (total_passwords + blockSize * batch_size - 1) / (blockSize * batch_size);

            int *d_found_flags;
            long long *d_result_indices;
            unsigned char *d_checked_bitmap;
            unsigned long long *d_global_start_index;

            hipMalloc(&d_found_flags, sizeof(int));
            hipMalloc(&d_result_indices, sizeof(long long));
            hipMalloc(&d_checked_bitmap, (total_passwords + 7) / 8);
            hipMalloc(&d_global_start_index, sizeof(unsigned long long));

            int found = 0;
            hipMemset(d_found_flags, 0, sizeof(int));
            hipMemset(d_checked_bitmap, 0, (total_passwords + 7) / 8);
            unsigned long long global_start_index = 0;
            hipMemcpy(d_global_start_index, &global_start_index, sizeof(unsigned long long), hipMemcpyHostToDevice);


            // Allocate and copy salt and target hash to device
            char *d_salt;
            uint8_t *d_target_hash;
            hipMalloc(&d_salt, salt_length * sizeof(char));
            hipMalloc(&d_target_hash, 32 * sizeof(uint8_t));
            hipMemcpy(d_salt, target_salt, salt_length * sizeof(char), hipMemcpyHostToDevice);
            hipMemcpy(d_target_hash, target_hash, 32 * sizeof(uint8_t), hipMemcpyHostToDevice);

            // Start timing
            auto start_time = std::chrono::high_resolution_clock::now();
            
            // Launch kernel
            find_passwords_optimized_multi<<<numBlocks, blockSize>>>(
                d_salt, d_target_hash, 1, d_found_flags, d_result_indices,
                d_checked_bitmap, d_global_start_index, batch_size);

            hipDeviceSynchronize();

            // Copy results back to host
            long long result_index;
            hipMemcpy(&found, d_found_flags, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&result_index, d_result_indices, sizeof(long long), hipMemcpyDeviceToHost);

            // End timing
            auto end_time = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> elapsed_seconds = end_time - start_time;

            if (found == 1)
            {
                std::cout << "Password found at index: " << result_index << "\n";
            }
            else
            {
                std::cout << "Password not found\n";
            }

            // Calculate GH/s
            double hashes_per_second = total_passwords / elapsed_seconds.count();
            double gigahashes_per_second = hashes_per_second / 1e9;
            std::cout << "Performance: " << gigahashes_per_second << " GH/s" << std::endl;

            // Free device memory
            hipFree(d_found_flags);
            hipFree(d_result_indices);
            hipFree(d_checked_bitmap);
            hipFree(d_global_start_index);
            hipFree(d_salt);
            hipFree(d_target_hash);
        }

        infile.close();
        return 0;
}