#include <iostream>
#include <cstring>
#include <sstream>
#include <iomanip>
#include <hip/hip_runtime.h>

#ifndef SHA256_CUH
#define SHA256_CUH

#include <string>

// Define charset as a constant array in GPU memory for password generation
__constant__ char charset[] = "abcd";
const int base = 4;  // Adjust to match the charset length
const int password_length = 3;

__device__ void generate_password_from_id(unsigned long long id, char *password) {
    for (int i = password_length - 1; i >= 0; --i) {
        // Access the charset from constant memory
        password[i] = charset[id % base];
        id /= base;
    }
    password[password_length] = '\0';  // Null-terminate the string
}

__global__ void generate_passwords_kernel(char *output, int total_ids) {
    unsigned long long id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= total_ids) return;

    char password[password_length + 1];
    generate_password_from_id(id, password);

    // Store this password in the output array
    int idx = id * (password_length + 1);
    for (int i = 0; i <= password_length; ++i) {
        output[idx + i] = password[i];
    }
}

// SHA256 constants for device
__constant__ static const uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// SHA256 host-side equivalent constants
static const uint32_t K_host[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

class SHA256 {

public:
    __device__ __host__ SHA256() {
        m_blocklen = 0;
        m_bitlen = 0;
        m_state[0] = 0x6a09e667;
        m_state[1] = 0xbb67ae85;
        m_state[2] = 0x3c6ef372;
        m_state[3] = 0xa54ff53a;
        m_state[4] = 0x510e527f;
        m_state[5] = 0x9b05688c;
        m_state[6] = 0x1f83d9ab;
        m_state[7] = 0x5be0cd19;
    }

    // Update for std::string is host-only
    __host__ void update(const std::string &data) {
        update(reinterpret_cast<const uint8_t*> (data.c_str()), data.size());
    }

    __device__ __host__ void update(const uint8_t* data, size_t length) {
        for (size_t i = 0; i < length; i++) {
            m_data[m_blocklen++] = data[i];
            if (m_blocklen == 64) {
                transform();
                m_bitlen += 512;
                m_blocklen = 0;
            }
        }
    }

    __device__ __host__ uint8_t* digest() {
        auto hash = new uint8_t[32];
        pad();
        revert(hash);
        return hash;
    }

    // Host-only function for converting hash to hex string
    __host__ static std::string toString(const uint8_t* digest) {
        std::stringstream s;
        s << std::setfill('0') << std::hex;
        for (uint8_t i = 0; i < 32; i++) {
            s << std::setw(2) << (unsigned int)digest[i];
        }
        return s.str();
    }

private:
    uint8_t m_data[64];
    uint32_t m_blocklen;
    uint64_t m_bitlen;
    uint32_t m_state[8]; // A, B, C, D, E, F, G, H

    __device__ __host__ static uint32_t rotr(uint32_t x, uint32_t n) {
        return (x >> n) | (x << (32 - n));
    }

    __device__ __host__ static uint32_t choose(uint32_t e, uint32_t f, uint32_t g) {
        return (e & f) ^ (~e & g);
    }

    __device__ __host__ static uint32_t majority(uint32_t a, uint32_t b, uint32_t c) {
        return (a & (b | c)) | (b & c);
    }

    __device__ __host__ static uint32_t sig0(uint32_t x) {
        return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
    }

    __device__ __host__ static uint32_t sig1(uint32_t x) {
        return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
    }

    __device__ __host__ void transform() {
        uint32_t maj, xorA, ch, xorE, sum, newA, newE, m[64];
        uint32_t state[8];

        // Select K array based on whether we're on the device or host
#ifdef __CUDA_ARCH__
        const uint32_t* k_array = K; // For device
#else
        const uint32_t* k_array = K_host; // For host
#endif

        // Process the message schedule array (W)
        for (uint8_t i = 0, j = 0; i < 16; i++, j += 4) {
            m[i] = (m_data[j] << 24) | (m_data[j + 1] << 16) | (m_data[j + 2] << 8) | m_data[j + 3];
        }

        for (uint8_t k = 16; k < 64; k++) {
            m[k] = sig1(m[k - 2]) + m[k - 7] + sig0(m[k - 15]) + m[k - 16];
        }

        // Initialize state array with the current hash values
        for (uint8_t i = 0; i < 8; i++) {
            state[i] = m_state[i];
        }

        // Main compression loop
        for (uint8_t i = 0; i < 64; i++) {
            maj = majority(state[0], state[1], state[2]);
            xorA = rotr(state[0], 2) ^ rotr(state[0], 13) ^ rotr(state[0], 22);

            ch = choose(state[4], state[5], state[6]);
            xorE = rotr(state[4], 6) ^ rotr(state[4], 11) ^ rotr(state[4], 25);

            sum = m[i] + k_array[i] + state[7] + ch + xorE;
            newA = xorA + maj + sum;
            newE = state[3] + sum;

            state[7] = state[6];
            state[6] = state[5];
            state[5] = state[4];
            state[4] = newE;
            state[3] = state[2];
            state[2] = state[1];
            state[1] = state[0];
            state[0] = newA;
        }

        // Add the compressed chunk to the current hash value
        for (uint8_t i = 0; i < 8; i++) {
            m_state[i] += state[i];
        }
    }

    __device__ __host__ void pad() {
        uint64_t i = m_blocklen;
        uint8_t end = m_blocklen < 56 ? 56 : 64;

        m_data[i++] = 0x80; // Append 1 bit followed by zeros
        while (i < end) {
            m_data[i++] = 0x00;
        }

        if (m_blocklen >= 56) {
            transform();
            memset(m_data, 0, 56);
        }

        m_bitlen += m_blocklen * 8;
        m_data[63] = m_bitlen;
        m_data[62] = m_bitlen >> 8;
        m_data[61] = m_bitlen >> 16;
        m_data[60] = m_bitlen >> 24;
        m_data[59] = m_bitlen >> 32;
        m_data[58] = m_bitlen >> 40;
        m_data[57] = m_bitlen >> 48;
        m_data[56] = m_bitlen >> 56;
        transform();
    }

    __device__ __host__ void revert(uint8_t* hash) {
        for (uint8_t i = 0; i < 4; i++) {
            for (uint8_t j = 0; j < 8; j++) {
                hash[i + (j * 4)] = (m_state[j] >> (24 - i * 8)) & 0x000000ff;
            }
        }
    }
};

#endif

int main() {
    int total_passwords = base * base * base;  // 4^3 = 64

    // Allocate array to store the passwords
    char *d_output;
    hipMalloc(&d_output, total_passwords * (password_length + 1) * sizeof(char));

    int num_threads = 16;  // Adjusted for demonstration
    int num_blocks = (total_passwords + num_threads - 1) / num_threads;  // Ensure we cover all IDs

    // Launch the kernel
    generate_passwords_kernel<<<num_blocks, num_threads>>>(d_output, total_passwords);

    // Copy result back to the host
    char *passwords = new char[total_passwords * (password_length + 1)];
    hipMemcpy(passwords, d_output, total_passwords * (password_length + 1) * sizeof(char), hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < total_passwords; ++i) {
        std::cout << "Password " << i << ": " << (passwords + i * (password_length + 1)) << std::endl;
    }

    // Free memory
    hipFree(d_output);
    delete[] passwords;

    // Example usage of SHA256 for "aaaaaa"
    SHA256 sha;
    sha.update("aaaaaa");
    uint8_t* digest = sha.digest();
    std::cout << "SHA256 of 'aaaaaa': " << SHA256::toString(digest) << std::endl;
    delete[] digest;

    return 0;
}
