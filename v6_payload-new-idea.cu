#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>

#ifndef SHA256_CUH
#define SHA256_CUH

// Add these color definitions at the top
#define RED     "\033[31m"
#define GREEN   "\033[32m"
#define YELLOW  "\033[33m"
#define BLUE    "\033[34m"
#define MAGENTA "\033[35m"
#define CYAN    "\033[36m"
#define RESET   "\033[0m"
#define BOLD    "\033[1m"

#define MAX_TARGETS 100
#define OPTIMAL_BLOCK_SIZE 256
#define BATCH_SIZE 1000

// __constant__ array for device-side K values
__constant__ static const uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__constant__ char d_target_salt[16 + 1];
__constant__ uint8_t d_target_hash[32];

// Host-side equivalent of K for use in host functions
static const uint32_t K_host[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

// Custom device-compatible string copy function
__device__ void cuda_strcpy(char *dest, const char *src, size_t max_length)
{
    size_t i = 0;
    while (src[i] && i < max_length - 1)
    { // Ensure we don't exceed max_length
        dest[i] = src[i];
        i++;
    }
    dest[i] = '\0'; // Null terminate
}

// Custom device-compatible string concatenate function
__device__ void cuda_strcat(char *dest, const char *src)
{
    while (*dest)
        dest++; // Move pointer to the end of dest
    while (*src)
    {
        *dest++ = *src++;
    }
    *dest = '\0'; // Null terminate
}

// Custom device-compatible string length function
__device__ size_t cuda_strlen(const char *str)
{
    size_t len = 0;
    while (*str++)
        len++;
    return len;
}

class SHA256
{
public:
    uint32_t m_state[8]; // A, B, C, D, E, F, G, H
    uint32_t m_saltState[8]; // State after salt processing
    uint64_t m_saltBitlen; // Bit length after salt processing
    uint32_t m_saltBlocklen; // Block length after salt processing

    __device__ __host__ SHA256()
    {
        reset();
    }

    __device__ __host__ void reset()
    {
        m_blocklen = 0;
        m_bitlen = 0;
        m_state[0] = 0x6a09e667;
        m_state[1] = 0xbb67ae85;
        m_state[2] = 0x3c6ef372;
        m_state[3] = 0xa54ff53a;
        m_state[4] = 0x510e527f;
        m_state[5] = 0x9b05688c;
        m_state[6] = 0x1f83d9ab;
        m_state[7] = 0x5be0cd19;
    }

    __device__ __host__ void initWithSalt(const uint8_t *salt, size_t salt_length)
    {
        reset();
        update(salt, salt_length);

        //Store the state after processing with the salt
        for (int i = 0; i < 8; i++)
        {
            m_saltState[i] = m_state[i];
        }
        m_saltBitlen = m_bitlen;
        m_saltBlocklen = m_blocklen;
    }  

    __device__ __host__ void resetToSaltState()
    {
        for (int i = 0; i < 8; i++)
        {
            m_state[i] = m_saltState[i];
        }
        m_bitlen = m_saltBitlen;
        m_blocklen = m_saltBlocklen;
    }

    __device__ __host__ void update(const uint8_t *data, size_t length)
    {
        for (size_t i = 0; i < length; i++)
        {
            m_data[m_blocklen++] = data[i];
            if (m_blocklen == 64)
            {
                transform();
                m_bitlen += 512;
                m_blocklen = 0;
            }
        }
    }

    __device__ __host__ void digest(uint8_t *hash)
    {
        pad();
        revert(hash);
    }

private:
    uint8_t m_data[64];
    uint32_t m_blocklen;
    uint64_t m_bitlen;

    __device__ __host__ static uint32_t rotr(uint32_t x, uint32_t n)
    {
        return (x >> n) | (x << (32 - n));
    }

    __device__ __host__ static uint32_t choose(uint32_t e, uint32_t f, uint32_t g)
    {
        return (e & f) ^ (~e & g);
    }

    __device__ __host__ static uint32_t majority(uint32_t a, uint32_t b, uint32_t c)
    {
        return (a & (b | c)) | (b & c);
    }

    __device__ __host__ static uint32_t sig0(uint32_t x)
    {
        return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
    }

    __device__ __host__ static uint32_t sig1(uint32_t x)
    {
        return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
    }

    __device__ __host__ void transform()
    {
        uint32_t maj, xorA, ch, xorE, sum, newA, newE, m[64];
        uint32_t state[8];

        #ifdef __CUDA_ARCH__
        const uint32_t *k_array = K;
        #else
        const uint32_t *k_array = K_host;
        #endif

        // Unroll the first loop for processing the message schedule array
        #pragma unroll 16
        for (uint8_t i = 0, j = 0; i < 16; i++, j += 4)
        {
            m[i] = (m_data[j] << 24) | (m_data[j + 1] << 16) | (m_data[j + 2] << 8) | m_data[j + 3];
        }

        // Unroll the second loop for the message schedule array
        #pragma unroll 48
        for (uint8_t k = 16; k < 64; k++)
        {
            m[k] = sig1(m[k - 2]) + m[k - 7] + sig0(m[k - 15]) + m[k - 16];
        }

        // Initialize state array with the current hash values
        #pragma unroll 8
        for (uint8_t i = 0; i < 8; i++)
        {
            state[i] = m_state[i];
        }

        // Main compression loop - fully unroll
        #pragma unroll 64
        for (uint8_t i = 0; i < 64; i++)
        {
            maj = majority(state[0], state[1], state[2]);
            xorA = rotr(state[0], 2) ^ rotr(state[0], 13) ^ rotr(state[0], 22);

            ch = choose(state[4], state[5], state[6]);
            xorE = rotr(state[4], 6) ^ rotr(state[4], 11) ^ rotr(state[4], 25);

            sum = m[i] + k_array[i] + state[7] + ch + xorE;
            newA = xorA + maj + sum;
            newE = state[3] + sum;

            state[7] = state[6];
            state[6] = state[5];
            state[5] = state[4];
            state[4] = newE;
            state[3] = state[2];
            state[2] = state[1];
            state[1] = state[0];
            state[0] = newA;
        }

        // Add the compressed chunk to the current hash value
        #pragma unroll 8
        for (uint8_t i = 0; i < 8; i++)
        {
            m_state[i] += state[i];
        }
    }

    __device__ __host__ void pad()
    {
        uint64_t i = m_blocklen;
        uint8_t end = m_blocklen < 56 ? 56 : 64;

        m_data[i++] = 0x80; // Append 1 bit followed by zeros
        while (i < end)
        {
            m_data[i++] = 0x00;
        }

        if (m_blocklen >= 56)
        {
            transform();
            memset(m_data, 0, 56);
        }

        m_bitlen += m_blocklen * 8;
        m_data[63] = m_bitlen;
        m_data[62] = m_bitlen >> 8;
        m_data[61] = m_bitlen >> 16;
        m_data[60] = m_bitlen >> 24;
        m_data[59] = m_bitlen >> 32;
        m_data[58] = m_bitlen >> 40;
        m_data[57] = m_bitlen >> 48;
        m_data[56] = m_bitlen >> 56;
        transform();
    }

    __device__ __host__ void revert(uint8_t *hash)
    {
        for (uint8_t i = 0; i < 4; i++)
        {
            for (uint8_t j = 0; j < 8; j++)
            {
                hash[i + (j * 4)] = (m_state[j] >> (24 - i * 8)) & 0x000000ff;
            }
        }
    }
};

#endif

__device__ void computeHash(const char *password, uint8_t *hashOutput)
{
    // Create an instance of SHA256
    SHA256 sha256;

    // Hash the password
    sha256.update((const uint8_t *)password, cuda_strlen(password));

    // Get the resulting hash
    sha256.digest(hashOutput);
}


__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
const int charset_size = 62; // Length of charset
const size_t password_length = 6;
const size_t salt_length = 16;


void hexToBytes(const char *hexString, uint8_t *byteArray)
{
    for (size_t i = 0; i < 32; ++i)
    {
        sscanf(hexString + 2 * i, "%2hhx", &byteArray[i]);
    }
}

__device__ void generate_password(long long idx, char *password)
{
    for (int i = 0; i < password_length; ++i)
    {
        password[i] = charset[idx % charset_size];
        idx /= charset_size;
    }
    password[password_length] = '\0'; // Null-terminate the string
}


__device__ bool compareUint8Arrays(const uint8_t* array1, const uint8_t* array2, size_t length) {
    for (size_t i = 0; i < length; ++i) {
        if (array1[i] != array2[i]) {
            return false; // Arrays differ at this position
        }
    }
    return true; // Arrays are identical
}

__global__ void find_passwords_optimized_multi(
    const char* salt,                
    const uint8_t* target_hashes,    
    int num_target_hashes,           
    int* found_flags,                
    long long* result_indices,       
    unsigned long long* global_start_index,   
    int batch_size,
    unsigned long long lowest_unfound_index  
) {
    // Shared memory for storing the initial SHA256 state with salt
    __shared__ SHA256 shared_sha256;

    // Initialize the SHA256 state with salt (only first thread in block)
    if (threadIdx.x == 0) {
        shared_sha256.initWithSalt((const uint8_t*)salt, salt_length);
    }

    __syncthreads(); // Ensure all threads have access to initialized shared memory

    // Get the starting index for this thread's batch of passwords
    long long thread_start_index = atomicAdd((unsigned long long*)global_start_index, (unsigned long long)batch_size);
    
    // Skip if below lowest_unfound_index
    if (thread_start_index < lowest_unfound_index) {
        thread_start_index = lowest_unfound_index;
    }
    
    uint8_t hash[32]; // Buffer to store computed hash

    // Initialize SHA256 object for this thread
    SHA256 sha256 = shared_sha256;

    // Process batch_size number of passwords
    for (int i = 0; i < batch_size; i++) {
        long long idx = thread_start_index + i;
        
        // Generate password for this index
        char password[password_length + 1];
        generate_password(idx, password);

        // Compute hash for the password
        sha256.resetToSaltState();
        sha256.update((const uint8_t*)password, password_length);
        sha256.digest(hash);

        // Compare with all target hashes
        for (int j = 0; j < num_target_hashes; j++) {
            if (!found_flags[j] && compareUint8Arrays(hash, target_hashes + j * 32, 32)) {
                // Atomically set the found flag and store the result index
                int old = atomicExch(&found_flags[j], 1);
                if (old == 0) {
                    result_indices[j] = idx;
                }
            }
        }
    }
}



__device__ int cuda_strcmp(const char* str1, const char* str2) {
    while (*str1 && (*str1 == *str2)) {
        str1++;
        str2++;
    }
    return *(const unsigned char*)str1 - *(const unsigned char*)str2;
}


//Test Code
void test_specific_case() {
    const char* test_password = "DNfOqt";
    const char* test_salt = "3ea1b3bbe71c269e";
    const char* expected_hash = "2f0a37f034bbd96a50f62b0b861748df53d6bde4e4926b6a57befdcb51efb8c6";
    
    printf("%s%s=== SHA256 Hash Test ===%s\n", BOLD, CYAN, RESET);
    printf("%s▶ Input Password:%s %s\n", YELLOW, RESET, test_password);
    printf("%s▶ Input Salt:%s %s\n", YELLOW, RESET, test_salt);
    printf("%s▶ Expected Hash:%s %s\n", YELLOW, RESET, expected_hash);

    // Create SHA256 instance
    SHA256 sha256;
    
    // Process salt
    sha256.update((const uint8_t*)test_salt, strlen(test_salt));
    
    // Process password
    sha256.update((const uint8_t*)test_password, strlen(test_password));
    
    // Get final hash
    uint8_t hash[32];
    sha256.digest(hash);
    
    printf("%s▶ Computed Hash:%s ", YELLOW, RESET);
    for(int i = 0; i < 32; i++) {
        printf("%02x", hash[i]);
    }
    printf("\n");

    // Compare hashes
    char computed_hash[65];
    for(int i = 0; i < 32; i++) {
        sprintf(&computed_hash[i*2], "%02x", hash[i]);
    }
    computed_hash[64] = '\0';

    if (strcmp(computed_hash, expected_hash) == 0) {
        printf("%s✓ Hash verification passed!%s\n\n", GREEN, RESET);
    } else {
        printf("%s✗ Hash verification failed!%s\n\n", RED, RESET);
    }
}


__global__ void test_password_generation() {
    long long test_idx = 3453345344623LL;
    char password[7];  // 6 chars + null terminator
    
    generate_password(test_idx, password);
    
    printf("%s%s=== Password Generation Test ===%s\n", BOLD, CYAN, RESET);
    printf("%s▶ Input index:%s %lld\n", YELLOW, RESET, test_idx);
    printf("%s▶ Generated:%s %s\n", YELLOW, RESET, password);
    printf("%s▶ Expected:%s PziZDX\n", YELLOW, RESET);

    if (cuda_strcmp(password, "PziZDX") == 0) {
        printf("%s✓ Test passed!%s\n\n", GREEN, RESET);
    } else {
        printf("%s✗ Test failed!%s\n\n", RED, RESET);
    }
}

// Test salt processing
__global__ void test_salt_processing() {
    printf("%s%s=== Salt Processing Test ===%s\n", BOLD, CYAN, RESET);
    
    const char* test_salt = "3ea1b3bbe71c269e";
    
    // Expected values after salt processing for "3ea1b3bbe71c269e"
    const uint32_t expected_state[8] = {
        0x6a09e667, // Initial state 0
        0xbb67ae85, // Initial state 1
        0x3c6ef372, // Initial state 2
        0xa54ff53a, // Initial state 3
        0x510e527f, // Initial state 4
        0x9b05688c, // Initial state 5
        0x1f83d9ab, // Initial state 6
        0x5be0cd19  // Initial state 7
    };
    
    SHA256 sha256;
    sha256.initWithSalt((const uint8_t*)test_salt, 16);
    
    printf("%s▶ Testing Salt State:%s\n", YELLOW, RESET);
    bool all_correct = true;
    for(int i = 0; i < 8; i++) {
        bool state_match = (sha256.m_saltState[i] == expected_state[i]);
        printf("State[%d]: Current=%08x, Expected=%08x %s%s%s\n", 
            i, 
            sha256.m_saltState[i], 
            expected_state[i],
            state_match ? GREEN : RED,
            state_match ? "✓" : "✗",
            RESET);
        all_correct &= state_match;
    }
    
    printf("\n%s▶ Final Result: %s%s%s\n", 
        YELLOW,
        all_correct ? GREEN : RED,
        all_correct ? "All states match!" : "States do not match!",
        RESET);
}


// Test bitmap tracking
__global__ void test_bitmap_tracking() {
    printf("%s%s=== Bitmap Tracking Test ===%s\n", BOLD, CYAN, RESET);
    
    unsigned char bitmap[8] = {0}; // Test with 64 bits
    
    // Test cases with large indices
    long long test_indices[] = {0LL, 7LL, 63LL};
    
    printf("%s▶ Testing Bit Operations:%s\n", YELLOW, RESET);
    
    for(int i = 0; i < 3; i++) {
        long long byte_index = test_indices[i] / 8;
        int bit_index = test_indices[i] % 8;
        unsigned int mask = 1U << bit_index;
        
        printf("\nTest %d: Index %lld\n", i + 1, test_indices[i]);
        printf("Byte Index: %lld, Bit Index: %d\n", byte_index, bit_index);
        
        unsigned char before = bitmap[byte_index];
        atomicOr((unsigned int*)&bitmap[byte_index], mask);
        unsigned char after = bitmap[byte_index];
        
        printf("Bitmap state - Before: 0x%02x, After: 0x%02x\n", before, after);
    }
}




// Test hash comparison
__global__ void test_hash_comparison() {
    printf("%s%s=== Hash Comparison Test ===%s\n", BOLD, CYAN, RESET);
    
    uint8_t hash1[32] = {0x2f, 0x0a, 0x37, 0xf0}; // First few bytes of our target hash
    uint8_t hash2[32] = {0x2f, 0x0a, 0x37, 0xf0}; // Matching hash
    uint8_t hash3[32] = {0x2f, 0x0a, 0x37, 0xf1}; // Different hash
    
    printf("%s▶ Testing matching hashes:%s\n", YELLOW, RESET);
    bool match1 = compareUint8Arrays(hash1, hash2, 32);
    printf("Match result: %s%s%s\n", match1 ? GREEN : RED, match1 ? "✓" : "✗", RESET);
    
    printf("%s▶ Testing different hashes:%s\n", YELLOW, RESET);
    bool match2 = compareUint8Arrays(hash1, hash3, 32);
    printf("Match result: %s%s%s\n", match2 ? GREEN : RED, match2 ? "✓" : "✗", RESET);
}

int main() {
    // GPU configuration for RTX 4090
    int maxThreadsPerBlock;
    int maxBlocksPerSM;
    int numSMs;

    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&maxBlocksPerSM, hipDeviceAttributeMaxBlocksPerMultiprocessor, 0);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    // Open input file
    std::ifstream infile("in.txt");
    if (!infile) {
        std::cerr << "Unable to open file in.txt";
        return 1;
    }

    std::string line;
    while (std::getline(infile, line)) {
        std::string salt_hex_string = line.substr(0, 16);
        std::string target_hash_string = line.substr(18, 64);

        printf("=== Processing New Hash ===\n");
        printf("▶ Salt: %s\n", salt_hex_string.c_str());
        printf("▶ Target Hash: %s\n", target_hash_string.c_str());

        const char *target_salt = salt_hex_string.c_str();
        const char *target_hash_hex = target_hash_string.c_str();
        uint8_t target_hash[32];
        hexToBytes(target_hash_hex, target_hash);

        // Calculate total passwords and optimal configuration
        unsigned long long total_passwords = 62ULL * 62 * 62 * 62 * 62 * 62; // 62^6
        int blockSize = 256;
        int batch_size = 1000;
        int numBlocks = numSMs * 32; // Optimal for RTX 4090

        // Allocate device memory
        int *d_found_flags;
        long long *d_result_indices;
        unsigned long long *d_global_start_index;
        char *d_salt;
        uint8_t *d_target_hash;

        hipMalloc(&d_found_flags, sizeof(int));
        hipMalloc(&d_result_indices, sizeof(long long));
        hipMalloc(&d_global_start_index, sizeof(unsigned long long));
        hipMalloc(&d_salt, salt_length * sizeof(char));
        hipMalloc(&d_target_hash, 32 * sizeof(uint8_t));

        // Initialize variables
        bool found = false;
        long long result_index = 0;
        unsigned long long processed_passwords = 0;
        
        // Start timing
        auto start_time = std::chrono::high_resolution_clock::now();

        // Process all password space
        while (!found && processed_passwords < total_passwords) {
            // Reset device memory for this batch
            hipMemset(d_found_flags, 0, sizeof(int));
            unsigned long long global_start_index = processed_passwords;
            hipMemcpy(d_global_start_index, &global_start_index, sizeof(unsigned long long), hipMemcpyHostToDevice);
            hipMemcpy(d_salt, target_salt, salt_length * sizeof(char), hipMemcpyHostToDevice);
            hipMemcpy(d_target_hash, target_hash, 32 * sizeof(uint8_t), hipMemcpyHostToDevice);

            // Launch kernel
            find_passwords_optimized_multi<<<numBlocks, blockSize>>>(
                d_salt, d_target_hash, 1, d_found_flags, d_result_indices,
                d_global_start_index, batch_size);

            hipDeviceSynchronize();

            // Check results
            int found_flag;
            hipMemcpy(&found_flag, d_found_flags, sizeof(int), hipMemcpyDeviceToHost);
            if (found_flag) {
                hipMemcpy(&result_index, d_result_indices, sizeof(long long), hipMemcpyDeviceToHost);
                found = true;
            }

            processed_passwords += (unsigned long long)numBlocks * blockSize * batch_size;
        }

        // Calculate performance
        auto end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end_time - start_time;
        double hashes_per_second = total_passwords / elapsed_seconds.count();
        double gigahashes_per_second = hashes_per_second / 1e9;


        // Print results with colors
        if (found) {
            printf("\033[1;32m✓ Password found at index: %lld\033[0m\n", result_index); // Bright green
        } else {
            printf("\033[1;31m✗ Password not found\033[0m\n"); // Bright red
        }
        printf("\033[1;33m▶ Performance: %.2f GH/s\033[0m\n", gigahashes_per_second); // Bright yellow


        // Free device memory
        hipFree(d_found_flags);
        hipFree(d_result_indices);
        hipFree(d_global_start_index);
        hipFree(d_salt);
        hipFree(d_target_hash);
    }

    infile.close();
    return 0;
}