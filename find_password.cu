#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
const int charset_size = 62; // Length of charset
const size_t password_length = 6;

__device__ void generate_password(long long idx, char* password) {
    for (int i = 0; i < password_length; ++i) {
        password[i] = charset[idx % charset_size];
        idx /= charset_size;
    }
    password[password_length] = '\0'; // Null-terminate the string
}

__device__ bool custom_strcmp(const char* a, const char* b) {
    for (int i = 0; i < password_length; ++i) {
        if (a[i] != b[i]) {
            return false;
        }
    }
    return true;
}

__global__ void find_password(long long start, long long end, const char* target_password, bool* found, long long* result_index) {
    long long idx = blockIdx.x * blockDim.x + threadIdx.x + start;

    if (idx < end) {
        char password[password_length + 1];
        generate_password(idx, password);

        if (custom_strcmp(password, target_password)) {
            *found = true;
            *result_index = idx;
        }
    }
}

int main() {
    const char* target_password = "D1hlVA";
    long long total_passwords = 62LL * 62 * 62 * 62 * 62 * 62; // 62^6 with explicit long long
    long long blockSize = 256; // Number of threads per block
    long long passwords_per_batch = 1000000; // Number of passwords to process in one batch
    long long num_batches = (total_passwords + passwords_per_batch - 1) / passwords_per_batch;

    char* d_target_password;
    bool* d_found;
    long long* d_result_index;

    hipMalloc(&d_target_password, (password_length + 1) * sizeof(char));
    hipMalloc(&d_found, sizeof(bool));
    hipMalloc(&d_result_index, sizeof(long long));

    hipMemcpy(d_target_password, target_password, (password_length + 1) * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_found, false, sizeof(bool));

    for (long long batch = 0; batch < num_batches; ++batch) {
        long long start = batch * passwords_per_batch;
        long long end = min(start + passwords_per_batch, total_passwords);

        // Calculate number of blocks needed for this batch
        long long numBlocks = (end - start + blockSize - 1) / blockSize;

        // Launch kernel for the current batch
        find_password<<<numBlocks, blockSize>>>(start, end, d_target_password, d_found, d_result_index);

        // Copy results back to host
        bool found;
        long long result_index;
        hipMemcpy(&found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(&result_index, d_result_index, sizeof(long long), hipMemcpyDeviceToHost);

        if (found) {
            std::cout << "Password found at index: " << result_index << "\n";
            break; // Exit loop if password is found
        }
    }

    // Free device memory
    hipFree(d_target_password);
    hipFree(d_found);
    hipFree(d_result_index);

    return 0;
}
