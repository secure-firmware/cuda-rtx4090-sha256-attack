#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>

// Define charset for password generation
__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
const int base = 62; // Charset length (lower + upper + numbers)
const int password_length = 6;
const int salt_length = 16;
const char salt[] = "671ddddb8aa8eec9"; // The example salt

// Predefined hash we're trying to match
const char predefined_hash_hex[] = "922482f6e20e95a35a3d150860bb1f03003c4f264e1dc7377e91b10eac5f8f10";

// SHA256 constants
__constant__ static const uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

// Custom device-compatible string copy function
__device__ void cuda_strcpy(char *dest, const char *src)
{
    while (*src)
    {
        *dest++ = *src++;
    }
    *dest = '\0'; // Null terminate
}

// Custom device-compatible string concatenate function
__device__ void cuda_strcat(char *dest, const char *src)
{
    while (*dest)
        dest++; // Move pointer to the end of dest
    while (*src)
    {
        *dest++ = *src++;
    }
    *dest = '\0'; // Null terminate
}

// Custom device-compatible string length function
__device__ size_t cuda_strlen(const char *str)
{
    size_t len = 0;
    while (*str++)
        len++;
    return len;
}

// SHA256 utility functions
__device__ __host__ static uint32_t rotr(uint32_t x, uint32_t n)
{
    return (x >> n) | (x << (32 - n));
}

__device__ __host__ static uint32_t choose(uint32_t e, uint32_t f, uint32_t g)
{
    return (e & f) ^ (~e & g);
}

__device__ __host__ static uint32_t majority(uint32_t a, uint32_t b, uint32_t c)
{
    return (a & (b | c)) | (b & c);
}

__device__ __host__ static uint32_t sig0(uint32_t x)
{
    return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
}

__device__ __host__ static uint32_t sig1(uint32_t x)
{
    return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
}

// SHA256 class definition
class SHA256
{
public:
    __device__ __host__ SHA256()
    {
        m_blocklen = 0;
        m_bitlen = 0;
        m_state[0] = 0x6a09e667;
        m_state[1] = 0xbb67ae85;
        m_state[2] = 0x3c6ef372;
        m_state[3] = 0xa54ff53a;
        m_state[4] = 0x510e527f;
        m_state[5] = 0x9b05688c;
        m_state[6] = 0x1f83d9ab;
        m_state[7] = 0x5be0cd19;
    }

    __device__ __host__ void update(const uint8_t *data, size_t length)
    {
        for (size_t i = 0; i < length; i++)
        {
            m_data[m_blocklen++] = data[i];
            if (m_blocklen == 64)
            {
                transform();
                m_bitlen += 512;
                m_blocklen = 0;
            }
        }
    }

    __device__ __host__ void finalize(uint8_t *hash)
    {
        pad();
        revert(hash);
    }

private:
    uint8_t m_data[64];
    uint32_t m_blocklen;
    uint64_t m_bitlen;
    uint32_t m_state[8]; // A, B, C, D, E, F, G, H

    __device__ __host__ void transform()
    {
        uint32_t maj, xorA, ch, xorE, sum, newA, newE, m[64];
        uint32_t state[8];

        for (uint8_t i = 0, j = 0; i < 16; i++, j += 4)
        {
            m[i] = (m_data[j] << 24) | (m_data[j + 1] << 16) | (m_data[j + 2] << 8) | m_data[j + 3];
        }

        for (uint8_t k = 16; k < 64; k++)
        {
            m[k] = sig1(m[k - 2]) + m[k - 7] + sig0(m[k - 15]) + m[k - 16];
        }

        for (uint8_t i = 0; i < 8; i++)
        {
            state[i] = m_state[i];
        }

        for (uint8_t i = 0; i < 64; i++)
        {
            maj = majority(state[0], state[1], state[2]);
            xorA = rotr(state[0], 2) ^ rotr(state[0], 13) ^ rotr(state[0], 22);
            ch = choose(state[4], state[5], state[6]);
            xorE = rotr(state[4], 6) ^ rotr(state[4], 11) ^ rotr(state[4], 25);
            sum = m[i] + K[i] + state[7] + ch + xorE;
            newA = xorA + maj + sum;
            newE = state[3] + sum;

            state[7] = state[6];
            state[6] = state[5];
            state[5] = state[4];
            state[4] = newE;
            state[3] = state[2];
            state[2] = state[1];
            state[1] = state[0];
            state[0] = newA;
        }

        for (uint8_t i = 0; i < 8; i++)
        {
            m_state[i] += state[i];
        }
    }

    __device__ __host__ void pad()
    {
        uint64_t i = m_blocklen;
        uint8_t end = m_blocklen < 56 ? 56 : 64;

        m_data[i++] = 0x80; // Append 1 bit followed by zeros
        while (i < end)
        {
            m_data[i++] = 0x00;
        }

        if (m_blocklen >= 56)
        {
            transform();
            memset(m_data, 0, 56);
        }

        m_bitlen += m_blocklen * 8;
        m_data[63] = m_bitlen;
        m_data[62] = m_bitlen >> 8;
        m_data[61] = m_bitlen >> 16;
        m_data[60] = m_bitlen >> 24;
        m_data[59] = m_bitlen >> 32;
        m_data[58] = m_bitlen >> 40;
        m_data[57] = m_bitlen >> 48;
        m_data[56] = m_bitlen >> 56;
        transform();
    }

    __device__ __host__ void revert(uint8_t *hash)
    {
        for (uint8_t i = 0; i < 4; i++)
        {
            for (uint8_t j = 0; j < 8; j++)
            {
                hash[i + (j * 4)] = (m_state[j] >> (24 - i * 8)) & 0x000000ff;
            }
        }
    }
};

// Convert a hex string to a byte array
__host__ void hex_to_bytes(const char *hex, uint8_t *bytes)
{
    for (int i = 0; i < 32; i++)
    {
        sscanf(hex + 2 * i, "%2hhx", &bytes[i]);
    }
}

__device__ __host__ void hash_to_string(const uint8_t *hash, char *output)
{
    const char hex_digits[] = "0123456789abcdef";

    for (int i = 0; i < 32; ++i)
    {
        output[i * 2] = hex_digits[(hash[i] >> 4) & 0x0F]; // Upper nibble
        output[i * 2 + 1] = hex_digits[hash[i] & 0x0F];    // Lower nibble
    }

    output[64] = '\0'; // Null-terminate the string
}

// Check if two byte arrays are equal
__device__ __host__ bool compare_hashes(const uint8_t *hash1, const uint8_t *hash2)
{
    for (int i = 0; i < 32; i++)
    {
        if (hash1[i] != hash2[i])
        {
            return false;
        }
    }
    return true;
}

__device__ int password_found = 0;

__global__ void brute_force_kernel(const char *salt, const uint8_t *target_hash, char *result, unsigned long long total_ids, uint8_t *temp_hash)
{
    unsigned long long id = blockIdx.x * blockDim.x + threadIdx.x;

    // If the password is already found, exit early
    if (password_found || id >= total_ids) return;

    char password[password_length + 1];
    char combined[password_length + salt_length + 1];
    uint8_t hash[32];

    unsigned long long temp_id = id;
    for (size_t i = 0; i < password_length; i++)
    {
        password[i] = charset[temp_id % base];
        temp_id /= base;
    }
    password[password_length] = '\0';
    cuda_strcpy(combined, password);
    cuda_strcat(combined, salt);

    // Hash the combined password+salt
    SHA256 sha;
    sha.update((uint8_t *)combined, cuda_strlen(combined));
    sha.finalize(hash);

    // Compare the hash to the target hash
    if (compare_hashes(hash, target_hash))
    {
        // Use atomicExch to ensure only one thread writes the result
        if (atomicExch(&password_found, 1) == 0)
        {
            cuda_strcpy(result, password);  // Copy the found password to result
            for (size_t i = 0; i < 32; i++)
            {
                temp_hash[i] = hash[i];  // Copy the found hash to temp_hash
            }
        }
    }
}

int main()
{

    uint8_t temp_hash[32];
    uint8_t *d_temp_hash;
    char temp_hex[65];
    hipMalloc(&d_temp_hash, sizeof(temp_hash));

    uint8_t target_hash[32];
    hex_to_bytes(predefined_hash_hex, target_hash);

    // Define the number of possible passwords (62^6)
    unsigned long long total_passwords = 1;
    for (int i = 0; i < password_length; i++)
    {
        total_passwords *= base;
    }

    // Allocate memory for the password and combined string
    char *d_salt;
    hipMalloc(&d_salt, salt_length + 1);
    hipMemcpy(d_salt, salt, salt_length + 1, hipMemcpyHostToDevice);

    char *d_result;
    hipMalloc(&d_result, password_length + salt_length + 1);
    hipMemset(d_result, 0, password_length + salt_length + 1);


    uint8_t *d_target_hash;
    hipMalloc(&d_target_hash, sizeof(target_hash));
    hipMemcpy(d_target_hash, target_hash, sizeof(target_hash), hipMemcpyHostToDevice);

    // Launch brute-force kernel
    int threads_per_block = 256;
    unsigned long long blocks_per_grid = (total_passwords + threads_per_block - 1) / threads_per_block;
    brute_force_kernel<<<blocks_per_grid, threads_per_block>>>(d_salt, d_target_hash, d_result, total_passwords, d_temp_hash);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }


    char result[password_length + salt_length + 1];
    hipMemcpy(result, d_result, password_length + salt_length + 1, hipMemcpyDeviceToHost);
    std::cout << "Result: " << result << std::endl;
    hipMemcpy(temp_hash, d_temp_hash, sizeof(temp_hash), hipMemcpyDeviceToHost);
    hash_to_string(temp_hash, temp_hex);

    std::cout << "Target Hash: " << predefined_hash_hex << std::endl;
    std::cout << "Temp hex: " << temp_hex << std::endl;

    hipFree(d_temp_hash);
    hipFree(d_salt);
    hipFree(d_result);
    hipFree(d_target_hash);
    return 0;
}