
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void sayHello() {
    printf("Hello, CUDA!\n");
}

int main() {
    sayHello<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
