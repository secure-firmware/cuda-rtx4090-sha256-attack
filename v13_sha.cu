#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include <vector>

#ifndef SHA256_CUH
#define SHA256_CUH

// Add these color definitions at the top
#define RED     "\033[31m"
#define GREEN   "\033[32m"
#define YELLOW  "\033[33m"
#define BLUE    "\033[34m"
#define MAGENTA "\033[35m"
#define CYAN    "\033[36m"
#define RESET   "\033[0m"
#define BOLD    "\033[1m"


__constant__ const unsigned long long total_passwords = 62ULL * 62 * 62 * 62 * 62 * 62;
__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
const int charset_size = 62; // Length of charset
const size_t password_length = 6;

// __constant__ array for device-side K values
__constant__ static const uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__constant__ char d_target_salt[16 + 1];
__constant__ uint8_t d_target_hash[32];

class SHA256 {
private:
    uint32_t m_state[8];
    uint8_t m_data[64];
    uint32_t m_blocklen;
    uint64_t m_bitlen;

    __device__ static uint32_t rotr(uint32_t x, uint32_t n) {
        return (x >> n) | (x << (32 - n));
    }

    __device__ static uint32_t choose(uint32_t e, uint32_t f, uint32_t g) {
        return (e & f) ^ (~e & g);
    }

    __device__ static uint32_t majority(uint32_t a, uint32_t b, uint32_t c) {
        return (a & (b | c)) | (b & c);
    }

    __device__ void transform() {
        uint32_t m[64];
        uint32_t a = m_state[0];
        uint32_t b = m_state[1];
        uint32_t c = m_state[2];
        uint32_t d = m_state[3];
        uint32_t e = m_state[4];
        uint32_t f = m_state[5];
        uint32_t g = m_state[6];
        uint32_t h = m_state[7];

        #pragma unroll 16
        for (uint8_t i = 0, j = 0; i < 16; i++, j += 4) {
            m[i] = (m_data[j] << 24) | (m_data[j + 1] << 16) | 
                   (m_data[j + 2] << 8) | m_data[j + 3];
        }

        #pragma unroll
        for(uint8_t i = 16; i < 64; i++) {
            uint32_t s0 = rotr(m[i-15], 7) ^ rotr(m[i-15], 18) ^ (m[i-15] >> 3);
            uint32_t s1 = rotr(m[i-2], 17) ^ rotr(m[i-2], 19) ^ (m[i-2] >> 10);
            m[i] = m[i-16] + s0 + m[i-7] + s1;
        }

        #pragma unroll
        for(uint8_t i = 0; i < 64; i++) {
            uint32_t S1 = rotr(e, 6) ^ rotr(e, 11) ^ rotr(e, 25);
            uint32_t ch = choose(e, f, g);
            uint32_t temp1 = h + S1 + ch + K[i] + m[i];
            uint32_t S0 = rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22);
            uint32_t maj = majority(a, b, c);
            uint32_t temp2 = S0 + maj;

            h = g;
            g = f;
            f = e;
            e = d + temp1;
            d = c;
            c = b;
            b = a;
            a = temp1 + temp2;
        }

        m_state[0] += a;
        m_state[1] += b;
        m_state[2] += c;
        m_state[3] += d;
        m_state[4] += e;
        m_state[5] += f;
        m_state[6] += g;
        m_state[7] += h;
    }

public:
    __device__ SHA256() {
        reset();
    }

    __device__ void reset() {
        m_blocklen = 0;
        m_bitlen = 0;
        m_state[0] = 0x6a09e667;
        m_state[1] = 0xbb67ae85;
        m_state[2] = 0x3c6ef372;
        m_state[3] = 0xa54ff53a;
        m_state[4] = 0x510e527f;
        m_state[5] = 0x9b05688c;
        m_state[6] = 0x1f83d9ab;
        m_state[7] = 0x5be0cd19;
    }

    __device__ void update(const uint8_t *data, size_t length) {
        for (size_t i = 0; i < length; i++) {
            m_data[m_blocklen++] = data[i];
            if (m_blocklen == 64) {
                transform();
                m_bitlen += 512;
                m_blocklen = 0;
            }
        }
    }

    __device__ void digest(uint8_t *hash) {
        uint64_t i = m_blocklen;
        uint8_t end = m_blocklen < 56 ? 56 : 64;

        m_data[i++] = 0x80;
        while (i < end) {
            m_data[i++] = 0x00;
        }

        if (m_blocklen >= 56) {
            transform();
            memset(m_data, 0, 56);
        }

        m_bitlen += m_blocklen * 8;
        m_data[63] = m_bitlen;
        m_data[62] = m_bitlen >> 8;
        m_data[61] = m_bitlen >> 16;
        m_data[60] = m_bitlen >> 24;
        m_data[59] = m_bitlen >> 32;
        m_data[58] = m_bitlen >> 40;
        m_data[57] = m_bitlen >> 48;
        m_data[56] = m_bitlen >> 56;
        transform();

        #pragma unroll
        for(uint8_t i = 0; i < 8; i++) {
            hash[i*4] = (m_state[i] >> 24) & 0xFF;
            hash[i*4 + 1] = (m_state[i] >> 16) & 0xFF;
            hash[i*4 + 2] = (m_state[i] >> 8) & 0xFF;
            hash[i*4 + 3] = m_state[i] & 0xFF;
        }
    }
};


#endif



void hexToBytes(const char *hexString, uint8_t *byteArray)
{
    for (size_t i = 0; i < 32; ++i)
    {
        sscanf(hexString + 2 * i, "%2hhx", &byteArray[i]);
    }
}

__device__ void generate_password(long long idx, char *password)
{
    for (int i = 0; i < password_length; ++i)
    {
        password[i] = charset[idx % charset_size];
        idx /= charset_size;
    }
    password[password_length] = '\0'; // Null-terminate the string
}

__device__ bool compareUint8Arrays(const uint8_t* array1, const uint8_t* array2, size_t length) {
    for (size_t i = 0; i < length; ++i) {
        if (array1[i] != array2[i]) {
            return false; // Arrays differ at this position
        }
    }
    return true; // Arrays are identical
}


__global__ void find_passwords_optimized_multi(
    const uint8_t* salt,                
    const uint8_t* target_hashes,    
    int num_target_hashes,           
    unsigned long long* global_start_index,   
    int batch_size,
    unsigned long long lowest_unfound_index  
) {
    long long base_index = lowest_unfound_index + blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < batch_size; i++) {
        long long idx = base_index + i * gridDim.x * blockDim.x;
        if (idx >= total_passwords) return;

        char password[password_length + 1];
        generate_password(idx, password);

        uint8_t hash[32];
        SHA256 sha256;
        sha256.update((const uint8_t*)password, password_length);
        sha256.update(salt, 8);
        sha256.digest(hash);

        for (int j = 0; j < num_target_hashes; j++) {
            if (compareUint8Arrays(hash, target_hashes + j * 32, 32)) {
                // Print in format: hash:salt:password (index: xxx)
                printf("%.2x%.2x%.2x...:%02x%02x%02x...:%s (index: %lld)\n", 
                    target_hashes[j * 32], target_hashes[j * 32 + 1], target_hashes[j * 32 + 2],
                    salt[0], salt[1], salt[2],
                    password, idx);
            }
        }
    }
}




int main() {
    int maxThreadsPerBlock;
    int maxBlocksPerSM;
    int numSMs;

    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&maxBlocksPerSM, hipDeviceAttributeMaxBlocksPerMultiprocessor, 0);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    const int MAX_HASHES = 100;
    struct HashPair {
        char salt[17];
        char hash[65];
    };
    HashPair all_hashes[MAX_HASHES];
    int num_hashes = 0;

    std::ifstream infile("in.txt");
    if (!infile) {
        std::cerr << "Unable to open file in.txt";
        return 1;
    }

    std::string line;
    while (std::getline(infile, line) && num_hashes < MAX_HASHES) {
        strncpy(all_hashes[num_hashes].salt, line.substr(65, 16).c_str(), 16);
        strncpy(all_hashes[num_hashes].hash, line.substr(0, 64).c_str(), 64);
        all_hashes[num_hashes].salt[16] = '\0';
        all_hashes[num_hashes].hash[64] = '\0';
        num_hashes++;
    }

    uint8_t all_target_hashes[MAX_HASHES * 32];
    uint8_t all_target_salts[MAX_HASHES * 8];
    
    for (int i = 0; i < num_hashes; i++) {
        hexToBytes(all_hashes[i].hash, &all_target_hashes[i * 32]);
        hexToBytes(all_hashes[i].salt, &all_target_salts[i * 8]);
    }

    uint8_t *d_target_salts;
    uint8_t *d_target_hashes;
    unsigned long long *d_global_start_index;

    hipMalloc(&d_target_salts, num_hashes * 8);
    hipMalloc(&d_target_hashes, num_hashes * 32);
    hipMalloc(&d_global_start_index, sizeof(unsigned long long));

    hipMemcpy(d_target_salts, all_target_salts, num_hashes * 8, hipMemcpyHostToDevice);
    hipMemcpy(d_target_hashes, all_target_hashes, num_hashes * 32, hipMemcpyHostToDevice);

    int blockSize = 512;
    int batch_size = 100;
    int numBlocks = numSMs * 32;
    unsigned long long lowest_unfound_index = 0;

    auto start_time = std::chrono::high_resolution_clock::now();

    while (lowest_unfound_index < total_passwords) {
        find_passwords_optimized_multi<<<numBlocks, blockSize>>>(
            d_target_salts,
            d_target_hashes,
            num_hashes,
            d_global_start_index,
            batch_size,
            lowest_unfound_index
        );
        hipDeviceSynchronize();
        lowest_unfound_index += numBlocks * blockSize * batch_size;
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = end_time - start_time;

    printf("\nTotal time: %.2f seconds\n", elapsed_seconds.count());
    printf("Performance: %.2f GH/s\n", total_passwords / elapsed_seconds.count() / 1e9);

    hipFree(d_target_salts);
    hipFree(d_target_hashes);
    hipFree(d_global_start_index);

    return 0;
}
