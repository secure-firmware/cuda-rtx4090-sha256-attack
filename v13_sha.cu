#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include <vector>

#ifndef SHA256_CUH
#define SHA256_CUH

// Add these color definitions at the top
#define RED     "\033[31m"
#define GREEN   "\033[32m"
#define YELLOW  "\033[33m"
#define BLUE    "\033[34m"
#define MAGENTA "\033[35m"
#define CYAN    "\033[36m"
#define RESET   "\033[0m"
#define BOLD    "\033[1m"


__constant__ const unsigned long long total_passwords = 62ULL * 62 * 62 * 62 * 62 * 62;
__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
const int charset_size = 62; // Length of charset
const size_t password_length = 6;

// __constant__ array for device-side K values
__constant__ static const uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__constant__ char d_target_salt[16 + 1];
__constant__ uint8_t d_target_hash[32];

class SHA256 {
private:
    uint32_t m_state[8];
    uint8_t m_data[64];

    __device__ static uint32_t rotr(uint32_t x, uint32_t n) {
        return (x >> n) | (x << (32 - n));
    }

    __device__ void transform() {
        uint32_t W[64];  // Message schedule array
        uint32_t a = m_state[0];
        uint32_t b = m_state[1];
        uint32_t c = m_state[2];
        uint32_t d = m_state[3];
        uint32_t e = m_state[4];
        uint32_t f = m_state[5];
        uint32_t g = m_state[6];
        uint32_t h = m_state[7];

        // Initial 16 words setup
        W[0] = ((uint32_t)m_data[0] << 24) | ((uint32_t)m_data[1] << 16) | 
               ((uint32_t)m_data[2] << 8) | m_data[3];
        W[1] = ((uint32_t)m_data[4] << 24) | ((uint32_t)m_data[5] << 16);
        W[2] = ((uint32_t)m_data[6] << 24) | ((uint32_t)m_data[7] << 16) | 
               ((uint32_t)m_data[8] << 8) | m_data[9];
        W[3] = ((uint32_t)m_data[10] << 24) | ((uint32_t)m_data[11] << 16) | 
               ((uint32_t)m_data[12] << 8) | ((uint32_t)m_data[13] | 0x80);

        #pragma unroll 11
        for(int i = 4; i < 15; i++) {
            W[i] = 0;
        }
        W[15] = 112;  // 14 bytes * 8 bits

        // Message schedule expansion
        #pragma unroll 48
        for(int i = 16; i < 64; i++) {
            uint32_t s0 = rotr(W[i-15], 7) ^ rotr(W[i-15], 18) ^ (W[i-15] >> 3);
            uint32_t s1 = rotr(W[i-2], 17) ^ rotr(W[i-2], 19) ^ (W[i-2] >> 10);
            W[i] = W[i-16] + s0 + W[i-7] + s1;
        }

        // Compression function
        #pragma unroll 64
        for(int i = 0; i < 64; i++) {
            uint32_t S1 = rotr(e, 6) ^ rotr(e, 11) ^ rotr(e, 25);
            uint32_t ch = (e & f) ^ (~e & g);
            uint32_t temp1 = h + S1 + ch + K[i] + W[i];
            uint32_t S0 = rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22);
            uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
            uint32_t temp2 = S0 + maj;

            h = g;
            g = f;
            f = e;
            e = d + temp1;
            d = c;
            c = b;
            b = a;
            a = temp1 + temp2;
        }

        m_state[0] += a;
        m_state[1] += b;
        m_state[2] += c;
        m_state[3] += d;
        m_state[4] += e;
        m_state[5] += f;
        m_state[6] += g;
        m_state[7] += h;
    }

public:
    __device__ SHA256() {
        reset();
    }

    __device__ void reset() {
        m_state[0] = 0x6a09e667;
        m_state[1] = 0xbb67ae85;
        m_state[2] = 0x3c6ef372;
        m_state[3] = 0xa54ff53a;
        m_state[4] = 0x510e527f;
        m_state[5] = 0x9b05688c;
        m_state[6] = 0x1f83d9ab;
        m_state[7] = 0x5be0cd19;
    }

    __device__ void update(const uint8_t *data, size_t length) {
        #pragma unroll
        for (size_t i = 0; i < length; i++) {
            m_data[i] = data[i];
        }
    }

    __device__ void digest(uint8_t *hash) {
        transform();
        
        #pragma unroll 8
        for(uint8_t i = 0; i < 8; i++) {
            hash[i*4] = (m_state[i] >> 24) & 0xFF;
            hash[i*4 + 1] = (m_state[i] >> 16) & 0xFF;
            hash[i*4 + 2] = (m_state[i] >> 8) & 0xFF;
            hash[i*4 + 3] = m_state[i] & 0xFF;
        }
    }
};




#endif

__device__ bool compareArrays(const uint8_t* arr1, const uint8_t* arr2, size_t length) {
    for (size_t i = 0; i < length; ++i) {
        if (arr1[i] != arr2[i]) {
            return false;
        }
    }
    return true;
}

__device__ void test_sha256() {
    const char* test_password = "jNdRTA";  // 6 bytes
    const uint8_t test_salt[8] = {0x0e, 0x8b, 0x22, 0xdf, 0xc5, 0x89, 0xe8, 0x7a}; // 8 bytes
    uint8_t hash[32];
    
    // Expected hash for "jNdRTA" with salt "0e8b22dfc589e87a"
    const uint8_t expected[32] = {
        0x82, 0x05, 0xde, 0x54, 0xcb, 0x32, 0x3e, 0x67,
        0xfb, 0x2c, 0x62, 0x74, 0xa2, 0xad, 0x4b, 0xd0,
        0x9c, 0xd8, 0x16, 0x24, 0xa0, 0x3b, 0x84, 0x82,
        0xfb, 0x61, 0x92, 0xee, 0x22, 0x16, 0x53, 0x2d
    };

    SHA256 sha256;
    sha256.update((const uint8_t*)test_password, 6);
    sha256.update(test_salt, 8);
    sha256.digest(hash);

    // Print results
    printf("Test password: %s\n", test_password);
    printf("Test salt: ");
    for(int i = 0; i < 8; i++) printf("%02x", test_salt[i]);
    printf("\nComputed hash: ");
    for(int i = 0; i < 32; i++) printf("%02x", hash[i]);
    printf("\nExpected hash: ");
    for(int i = 0; i < 32; i++) printf("%02x", expected[i]);
    printf("\nTest %s\n", compareArrays(hash, expected, 32) == 0 ? "PASSED" : "FAILED");
}




void hexToBytes(const char *hexString, uint8_t *byteArray)
{
    for (size_t i = 0; i < 32; ++i)
    {
        sscanf(hexString + 2 * i, "%2hhx", &byteArray[i]);
    }
}

__device__ void generate_password(long long idx, char *password)
{
    for (int i = 0; i < password_length; ++i)
    {
        password[i] = charset[idx % charset_size];
        idx /= charset_size;
    }
    password[password_length] = '\0'; // Null-terminate the string
}

__device__ bool compareUint8Arrays(const uint8_t* array1, const uint8_t* array2, size_t length) {
    for (size_t i = 0; i < length; ++i) {
        if (array1[i] != array2[i]) {
            return false; // Arrays differ at this position
        }
    }
    return true; // Arrays are identical
}


__global__ void find_passwords_optimized_multi(
    const uint8_t* salt,                
    const uint8_t* target_hashes,    
    int num_target_hashes,           
    unsigned long long* global_start_index,   
    int batch_size,
    unsigned long long lowest_unfound_index  
) {
    long long base_index = lowest_unfound_index + blockIdx.x * blockDim.x + threadIdx.x;

    if(base_index == 0) {
        test_sha256(); // Add test here
    }

    for (int i = 0; i < batch_size; i++) {
        long long idx = base_index + i * gridDim.x * blockDim.x;
        if (idx >= total_passwords) return;

        char password[password_length + 1];
        generate_password(idx, password);

        uint8_t hash[32];
        SHA256 sha256;
        sha256.update((const uint8_t*)password, password_length);
        sha256.update(salt, 8);
        sha256.digest(hash);

        for (int j = 0; j < num_target_hashes; j++) {
            if (compareUint8Arrays(hash, target_hashes + j * 32, 32)) {
                // Print in format: hash:salt:password (index: xxx)
                printf("%.2x%.2x%.2x...:%02x%02x%02x...:%s (index: %lld)\n", 
                    target_hashes[j * 32], target_hashes[j * 32 + 1], target_hashes[j * 32 + 2],
                    salt[0], salt[1], salt[2],
                    password, idx);
            }
        }
    }
}




int main() {
    int maxThreadsPerBlock;
    int maxBlocksPerSM;
    int numSMs;

    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&maxBlocksPerSM, hipDeviceAttributeMaxBlocksPerMultiprocessor, 0);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    const int MAX_HASHES = 100;
    struct HashPair {
        char salt[17];
        char hash[65];
    };
    HashPair all_hashes[MAX_HASHES];
    int num_hashes = 0;

    std::ifstream infile("in.txt");
    if (!infile) {
        std::cerr << "Unable to open file in.txt";
        return 1;
    }

    std::string line;
    while (std::getline(infile, line) && num_hashes < MAX_HASHES) {
        strncpy(all_hashes[num_hashes].salt, line.substr(65, 16).c_str(), 16);
        strncpy(all_hashes[num_hashes].hash, line.substr(0, 64).c_str(), 64);
        all_hashes[num_hashes].salt[16] = '\0';
        all_hashes[num_hashes].hash[64] = '\0';
        num_hashes++;
    }

    uint8_t all_target_hashes[MAX_HASHES * 32];
    uint8_t all_target_salts[MAX_HASHES * 8];
    
    for (int i = 0; i < num_hashes; i++) {
        hexToBytes(all_hashes[i].hash, &all_target_hashes[i * 32]);
        hexToBytes(all_hashes[i].salt, &all_target_salts[i * 8]);
    }

    uint8_t *d_target_salts;
    uint8_t *d_target_hashes;
    unsigned long long *d_global_start_index;

    hipMalloc(&d_target_salts, num_hashes * 8);
    hipMalloc(&d_target_hashes, num_hashes * 32);
    hipMalloc(&d_global_start_index, sizeof(unsigned long long));

    hipMemcpy(d_target_salts, all_target_salts, num_hashes * 8, hipMemcpyHostToDevice);
    hipMemcpy(d_target_hashes, all_target_hashes, num_hashes * 32, hipMemcpyHostToDevice);

    int blockSize = 512;
    int batch_size = 100;
    int numBlocks = numSMs * 32;
    unsigned long long lowest_unfound_index = 0;

    auto start_time = std::chrono::high_resolution_clock::now();

    while (lowest_unfound_index < total_passwords) {
        find_passwords_optimized_multi<<<numBlocks, blockSize>>>(
            d_target_salts,
            d_target_hashes,
            num_hashes,
            d_global_start_index,
            batch_size,
            lowest_unfound_index
        );
        hipDeviceSynchronize();
        lowest_unfound_index += numBlocks * blockSize * batch_size;
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = end_time - start_time;

    printf("\nTotal time: %.2f seconds\n", elapsed_seconds.count());
    printf("Performance: %.2f GH/s\n", total_passwords / elapsed_seconds.count() / 1e9);

    hipFree(d_target_salts);
    hipFree(d_target_hashes);
    hipFree(d_global_start_index);

    return 0;
}
