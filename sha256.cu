#include <iostream>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <cstring>

// SHA256 constants
__constant__ static const uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// Custom device-compatible string copy function
__device__ void cuda_strcpy(char* dest, const char* src) {
    while (*src) {
        *dest++ = *src++;
    }
    *dest = '\0';  // Null terminate
}

// Custom device-compatible string concatenate function
__device__ void cuda_strcat(char* dest, const char* src) {
    while (*dest) dest++;  // Move pointer to the end of dest
    while (*src) {
        *dest++ = *src++;
    }
    *dest = '\0';  // Null terminate
}

// Custom device-compatible string length function
__device__ size_t cuda_strlen(const char* str) {
    size_t len = 0;
    while (*str++) len++;
    return len;
}

// SHA256 utility functions
__device__ __host__ static uint32_t rotr(uint32_t x, uint32_t n) {
    return (x >> n) | (x << (32 - n));
}

__device__ __host__ static uint32_t choose(uint32_t e, uint32_t f, uint32_t g) {
    return (e & f) ^ (~e & g);
}

__device__ __host__ static uint32_t majority(uint32_t a, uint32_t b, uint32_t c) {
    return (a & (b | c)) | (b & c);
}

__device__ __host__ static uint32_t sig0(uint32_t x) {
    return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
}

__device__ __host__ static uint32_t sig1(uint32_t x) {
    return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
}

// SHA256 class definition
class SHA256 {
public:
    __device__ __host__ SHA256() {
        m_blocklen = 0;
        m_bitlen = 0;
        m_state[0] = 0x6a09e667;
        m_state[1] = 0xbb67ae85;
        m_state[2] = 0x3c6ef372;
        m_state[3] = 0xa54ff53a;
        m_state[4] = 0x510e527f;
        m_state[5] = 0x9b05688c;
        m_state[6] = 0x1f83d9ab;
        m_state[7] = 0x5be0cd19;
    }

    __device__ __host__ void update(const uint8_t* data, size_t length) {
        for (size_t i = 0; i < length; i++) {
            m_data[m_blocklen++] = data[i];
            if (m_blocklen == 64) {
                transform();
                m_bitlen += 512;
                m_blocklen = 0;
            }
        }
    }

    __device__ __host__ void finalize(uint8_t* hash) {
        pad();
        revert(hash);
    }

private:
    uint8_t  m_data[64];
    uint32_t m_blocklen;
    uint64_t m_bitlen;
    uint32_t m_state[8]; // A, B, C, D, E, F, G, H

    __device__ __host__ void transform() {
        uint32_t maj, xorA, ch, xorE, sum, newA, newE, m[64];
        uint32_t state[8];

        for (uint8_t i = 0, j = 0; i < 16; i++, j += 4) {
            m[i] = (m_data[j] << 24) | (m_data[j + 1] << 16) | (m_data[j + 2] << 8) | m_data[j + 3];
        }

        for (uint8_t k = 16; k < 64; k++) {
            m[k] = sig1(m[k - 2]) + m[k - 7] + sig0(m[k - 15]) + m[k - 16];
        }

        for(uint8_t i = 0 ; i < 8 ; i++) {
            state[i] = m_state[i];
        }

        for (uint8_t i = 0; i < 64; i++) {
            maj = majority(state[0], state[1], state[2]);
            xorA = rotr(state[0], 2) ^ rotr(state[0], 13) ^ rotr(state[0], 22);
            ch = choose(state[4], state[5], state[6]);
            xorE = rotr(state[4], 6) ^ rotr(state[4], 11) ^ rotr(state[4], 25);
            sum = m[i] + K[i] + state[7] + ch + xorE;
            newA = xorA + maj + sum;
            newE = state[3] + sum;

            state[7] = state[6];
            state[6] = state[5];
            state[5] = state[4];
            state[4] = newE;
            state[3] = state[2];
            state[2] = state[1];
            state[1] = state[0];
            state[0] = newA;
        }

        for(uint8_t i = 0 ; i < 8 ; i++) {
            m_state[i] += state[i];
        }
    }

    __device__ __host__ void pad() {
        uint64_t i = m_blocklen;
        uint8_t end = m_blocklen < 56 ? 56 : 64;

        m_data[i++] = 0x80; // Append 1 bit followed by zeros
        while (i < end) {
            m_data[i++] = 0x00;
        }

        if(m_blocklen >= 56) {
            transform();
            memset(m_data, 0, 56);
        }

        m_bitlen += m_blocklen * 8;
        m_data[63] = m_bitlen;
        m_data[62] = m_bitlen >> 8;
        m_data[61] = m_bitlen >> 16;
        m_data[60] = m_bitlen >> 24;
        m_data[59] = m_bitlen >> 32;
        m_data[58] = m_bitlen >> 40;
        m_data[57] = m_bitlen >> 48;
        m_data[56] = m_bitlen >> 56;
        transform();
    }

    __device__ __host__ void revert(uint8_t* hash) {
        for (uint8_t i = 0; i < 4; i++) {
            for(uint8_t j = 0; j < 8; j++) {
                hash[i + (j * 4)] = (m_state[j] >> (24 - i * 8)) & 0x000000ff;
            }
        }
    }
};

// Convert a hex string to a byte array
__host__ void hex_to_bytes(const char* hex, uint8_t* bytes) {
    for (int i = 0; i < 32; i++) {
        sscanf(hex + 2 * i, "%2hhx", &bytes[i]);
    }
}

int main() {

    return 0;
}
